#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

#include <assert.h>

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __device__ inline void dev_swap(int*& a, int*& b)
        {
            int* temp = a;
            a = b;
            b = temp;
        }

        __global__ void kernNaiveScan(int *w, int *r, int N, int stages, int* odata)
        {
            int k = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (k >= N)
                return;

            int offset = 1;
            for (int stage = 1; stage <= stages && offset < N; ++stage, offset <<= 1)
            {
                if (k >= offset)
                {
                    w[k] = r[k - offset] + r[k];
                }
                else
                {
                    w[k] = r[k];
                }

                dev_swap(w, r);
            }

            // At return, r contains the final result due to the swap
            odata[k] = r[k];
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            
            const int BLOCK_SIZE = 128;
            dim3 fullBlocksPerGrid((n + BLOCK_SIZE - 1) / BLOCK_SIZE);
            int stages = ilog2ceil(n);
            int N = 1 << stages; // next available power of two for N
            assert(stages == ilog2(N));

            // Alloc two device ping pong buffers, one for read and one for write
            int *dev_A, *dev_B;
            hipMalloc((void**)&dev_A, sizeof(int) * N);
            hipMalloc((void**)&dev_B, sizeof(int) * N);

            // A will be the read buffer for the first pass
            hipMemcpy((void*)dev_A, (const void*)idata, sizeof(int) * n, hipMemcpyHostToDevice);

            timer().startGpuTimer();         
            kernNaiveScan<<<fullBlocksPerGrid, BLOCK_SIZE>>>(dev_B, dev_A, N, stages, dev_B);
            timer().endGpuTimer();

            // Leave the first element empty for identity
            hipMemcpy((void*)&odata[1], (const void*)dev_B, sizeof(int) * (n-1), hipMemcpyDeviceToHost);

            // Identity
            odata[0] = 0;

            hipFree((void*)dev_A);
            hipFree((void*)dev_B);
        }
    }
}
